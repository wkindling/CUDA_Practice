#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define N1 2
#define N2 3
#define N3 2


using namespace std;


__global__ void multiple(float* A, float* B, float* C)
{
	int ix = threadIdx.x + blockDim.x*blockIdx.x;
	int iy = threadIdx.y + blockDim.y*blockIdx.y;
	int index = ix * N3 + iy;
	C[index] = 0;
	for (int i = 0; i < N2; i++)
	{
		C[index] += A[ix*N2 + i] * B[i*N3 + iy]; 		//C[ix,iy]+=A[ix,i]*B[i,iy]
	}
}

int main()
{
	hipSetDevice(0);
		
	float* A_host=(float*)malloc(N1*N2*sizeof(float));
	float* B_host=(float*)malloc(N2*N3*sizeof(float));
	float* C_host=(float*)malloc(N1*N3*sizeof(float));

	for (int i = 0; i < N1; i++)
	{
		for (int j = 0; j < N2; j++)
		{
			cin >> A_host[i*N2 + j];
		}
	}	
	
	for (int i = 0; i < N2; i++)
	{
		for (int j = 0; j < N3; j++)
		{
			cin >> B_host[i*N3 + j];
		}
	}

	float* A_dev = NULL;
	float* B_dev = NULL;
	float* C_dev = NULL;
	hipMalloc((void**)&A_dev, N1*N2 * sizeof(float));
	hipMalloc((void**)&B_dev, N2*N3 * sizeof(float));
	hipMalloc((void**)&C_dev, N1*N3 * sizeof(float));

	hipMemcpy(A_dev, A_host, sizeof(float)*N1*N2, hipMemcpyHostToDevice);
	hipMemcpy(B_dev, B_host, sizeof(float)*N2*N3, hipMemcpyHostToDevice);
	
	int dimx = N1;
	int dimy = N3;

	dim3 block(dimx, dimy);
	dim3 grid(1);

	multiple << <grid, block >> > (A_dev, B_dev, C_dev);
	
	hipDeviceSynchronize();
	hipMemcpy(C_host, C_dev, N1*N3 * sizeof(float), hipMemcpyDeviceToHost);

	cout <<endl;
	for (int i = 0; i < N1; i++)
	{
		for (int j = 0; j < N3; j++)
		{
			cout << C_host[i*N3+j] << " ";
		}
		cout << endl;
	}

	hipFree(A_dev);
	hipFree(B_dev);
	hipFree(C_dev);
	free(A_host);
	free(B_host);
	free(C_host);

	hipDeviceReset();
	return 0;
}