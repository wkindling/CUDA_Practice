#include "hip/hip_runtime.h"
//kernelVBO.cu

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>


__global__ void kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	float freq = 4.0f;

	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	pos[y*width + x] = make_float4(u, w, v, 1.0f);

}

extern "C" void launch_kernel(float4 *pos, unsigned int mesh_width, unsigned int mesh_height, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	kernel << <grid, block >> > (pos, mesh_width, mesh_height, time);

	hipDeviceSynchronize();
}